﻿#include <hip/hip_runtime.h>


#include <hip/device_functions.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <chrono>

#define ADDITION 1
#define MINIMUM 2
#define MAXIMUM 3

#define OPERATOR ADDITION	// can also be MINIMUM or MAXIMUM
#define ARR_SIZE 4194304

//Sample input sizes for testing:
/*
2^10 = 1024
2^12 = 4096
2^15 = 32769
2^17 = 131072
2^20 = 1048576
2^22 = 4194304
2^25 = 33554432
2^27 = 134217728
*/

//Blelloch's Parallel scan algorithm for large arrays that use multiple blocks
__global__ void blellochScanLarge(long long int* in, long long int* out, long long int* blockSums, long int arrSize) {
	__shared__ extern long long int myChunk[];
	
	int base = blockDim.x * 2 * blockIdx.x;
	int idx1 = threadIdx.x;
	int idx2 = threadIdx.x + blockDim.x;
	if (base + idx1 < arrSize)
		myChunk[idx1] = in[idx1 + base];
	else {
		if (OPERATOR == ADDITION)
			myChunk[idx1] = 0;
		else if (OPERATOR == MINIMUM)
			myChunk[idx1] = LLONG_MAX;
		else if (OPERATOR == MAXIMUM)
			myChunk[idx1] = LLONG_MIN;
	}
	if (base + idx2 < arrSize)
		myChunk[idx2] = in[idx2 + base];
	else {
		if (OPERATOR == ADDITION)
			myChunk[idx2] = 0;
		else if (OPERATOR == MINIMUM)
			myChunk[idx2] = LLONG_MAX;
		else if (OPERATOR == MAXIMUM)
			myChunk[idx2] = LLONG_MIN;
	}

	__syncthreads();

	//up-sweep
	for (int depth = 1; depth <= blockDim.x; depth <<= 1) {
		int dest = (threadIdx.x + 1) * 2 * depth - 1;
		if (dest < 2 * blockDim.x) {
			if (OPERATOR == ADDITION)
				myChunk[dest] += myChunk[dest - depth];
			else if (OPERATOR == MINIMUM)
				myChunk[dest] = myChunk[dest] > myChunk[dest - depth] ? myChunk[dest - depth] : myChunk[dest];
			else if (OPERATOR == MAXIMUM)
				myChunk[dest] = myChunk[dest] < myChunk[dest - depth] ? myChunk[dest - depth] : myChunk[dest];
		}
		__syncthreads();
	}

	//down-sweep
	for (int depth = blockDim.x >> 1; depth > 0; depth >>= 1) {
		int dest = (threadIdx.x + 1) * depth * 2 - 1;
		if (dest + depth < 2 * blockDim.x) {
			if (OPERATOR == ADDITION)
				myChunk[dest + depth] += myChunk[dest];
			else if(OPERATOR == MINIMUM)
				myChunk[dest + depth] = myChunk[dest + depth] > myChunk[dest] ? myChunk[dest] : myChunk[dest+depth];
			else if (OPERATOR == MAXIMUM)
				myChunk[dest + depth] = myChunk[dest + depth] < myChunk[dest] ? myChunk[dest] : myChunk[dest + depth];
		}
		__syncthreads();
	}

	if (base + idx1 < arrSize)
		out[idx1 + base] = myChunk[idx1];
	if (base + idx2 < arrSize)
		out[idx2 + base] = myChunk[idx2];

	__syncthreads();
	if (threadIdx.x == 0) {
		blockSums[blockIdx.x] = myChunk[2 * blockDim.x - 1];
	}
}

//Adds the cumulative block sums to all the elements in output array
__global__ void addBlockSums(long long int* out, long long int* blockSums, long int arrSize) {
	int threadId = 2 * blockDim.x * blockIdx.x + 2 * threadIdx.x;
	if (threadId + 1 < arrSize && blockIdx.x > 0) {
		if (OPERATOR == ADDITION)
			out[threadId + 1] += blockSums[blockIdx.x - 1];
		else if (OPERATOR == MINIMUM)
			out[threadId + 1] = out[threadId + 1] > blockSums[blockIdx.x - 1] ? blockSums[blockIdx.x - 1] : out[threadId + 1];
		else if (OPERATOR == MAXIMUM)
			out[threadId + 1] = out[threadId + 1] < blockSums[blockIdx.x - 1] ? blockSums[blockIdx.x - 1] : out[threadId + 1];
	
		if (OPERATOR == ADDITION)
			out[threadId] += blockSums[blockIdx.x - 1];
		else if (OPERATOR == MINIMUM)
			out[threadId] = out[threadId] > blockSums[blockIdx.x - 1] ? blockSums[blockIdx.x - 1] : out[threadId];
		else if (OPERATOR == MAXIMUM)
			out[threadId] = out[threadId] < blockSums[blockIdx.x - 1] ? blockSums[blockIdx.x - 1] : out[threadId];
	}
	else if (threadId < arrSize && blockIdx.x > 0) {
		if (OPERATOR == ADDITION)
			out[threadId] += blockSums[blockIdx.x - 1];
		else if (OPERATOR == MINIMUM)
			out[threadId] = out[threadId] > blockSums[blockIdx.x - 1] ? blockSums[blockIdx.x - 1] : out[threadId];
		else if (OPERATOR == MAXIMUM)
			out[threadId] = out[threadId] < blockSums[blockIdx.x - 1] ? blockSums[blockIdx.x - 1] : out[threadId];
	}
}

//Computes scan sequentially
void computeLinearScan(long long int* out, long long int* in, long int n) {
	out[0] = in[0];
	for (int i = 1; i < n; i++) {
		if (OPERATOR == ADDITION)
			out[i] = out[i - 1] + in[i];
		else if (OPERATOR == MINIMUM)
			out[i] = out[i - 1] > in[i] ? in[i] : out[i - 1];
		else if (OPERATOR == MAXIMUM)
			out[i] = out[i - 1] < in[i] ? in[i] : out[i - 1];
	}
}

//Verifies whether Parallel scan output is correct by comparing it with linear scan output
void verifyOutput(long long int* out1, long long int* out2, long int n) {
	int mismatches = 0;
	for (int i = 0; i < n; i++) {
		if (out1[i] != out2[i]) {
			if(mismatches == 0)
				printf("\nParallel scan first failed at i=%d. Serial output, Parallel Output : %lld, %lld", i, out1[i], out2[i]);
			mismatches++;
		}
	}
	if (mismatches == 0)
		printf("\nParallel scan output has been verified to be correct.\n");
	else
		printf("\nTotal of %d mismatches found between serial output and parallel output.\n", mismatches);
}

int main(int argc, char** argv) {

	srand(time(NULL));
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	hipDeviceProp_t pr;
	if (deviceCount == 0) {
		printf("No CUDA compatible GPU exists");
		return 0;
	}
	else {
		hipGetDeviceProperties(&pr, 0);
	}

	/*clock_t linearStart, parallelStart, linearEnd, parallelEnd, serialStart, serialEnd;*/
	//double linearTime, parallelTime, serialTime, masterTime;
	double parallelTime = 0.0;
	float elapsedTime1, elapsedTime2;
	elapsedTime1 = 0.0;
	elapsedTime2 = 0.0;

	/*
	linearTime = 0.0;
	parallelTime = 0.0;
	serialTime = 0.0;*/
	hipEvent_t start1, stop1, start2, stop2;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventCreate(&start2);
	hipEventCreate(&stop2);

	int threadsPerBlock = pr.maxThreadsPerBlock;
	int arrElementsPerBlock = 2 * threadsPerBlock;
	int blockCount = (int)ceil(((double)ARR_SIZE) / arrElementsPerBlock);

	//serialStart = clock();

	//host copies
	long long int* inArr, * outArr, * blockSums;
	//long long int* linearScanOutput;
	long long int allocSize1 = sizeof(long long) * (long)ARR_SIZE;
	long long int allocSize2 = sizeof(long long) * blockCount;
	long long int allocSize3 = sizeof(long long) * (2 * threadsPerBlock);

	//device copies
	long long int* devInArr, * devOutArr, * devBlockSums;

	//linearStart = clock();

	//linearScanOutput = (long long int*)malloc(allocSize1);
	inArr = (long long int*)malloc(allocSize1);
	outArr = (long long int*)malloc(allocSize1);
	blockSums = (long long int*)malloc(allocSize2);
	for (int i = 0; i < ARR_SIZE; i++) {
		//if (inArr != NULL)
			//inArr[i] = (rand() % 53) + (long long)17;
			inArr[i] = 1;
	}
	

	/*computeLinearScan(linearScanOutput, inArr, ARR_SIZE);
	serialEnd = clock();
	serialTime += (double)(serialEnd - serialStart);
	linearEnd = clock();*/
	//printf("\n%d, %d", linearStart, linearEnd);

	/*linearTime = (((double)(linearEnd - linearStart)) / (double)CLOCKS_PER_SEC) * 1000.00;

	parallelStart = clock();
	serialStart = clock();*/

	hipEventRecord(start1);
	hipMalloc((void**)&devInArr, allocSize1);
	hipMalloc((void**)&devOutArr, allocSize1);
	hipMalloc((void**)&devBlockSums, allocSize2);
	hipMemcpy(devInArr, inArr, allocSize1, hipMemcpyHostToDevice);
	/*serialEnd = clock();

	parallelEnd = clock();

	serialTime += (double)(serialEnd - serialStart);
	parallelTime += (((double)(parallelEnd - parallelStart)) / (double)CLOCKS_PER_SEC) * 1000.00;*/


	blellochScanLarge <<<blockCount, threadsPerBlock, allocSize3>>> (devInArr, devOutArr, devBlockSums, (long)ARR_SIZE);
	hipEventRecord(stop1);

	hipDeviceSynchronize();
	hipMemcpy(blockSums, devBlockSums, allocSize2, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&elapsedTime1, start1, stop1);

	/*serialStart = clock();
	parallelStart = clock();*/


	
	auto start = std::chrono::high_resolution_clock::now();

	for (int i = 1; i < blockCount; i++) {
		if (OPERATOR == ADDITION)
			blockSums[i] += blockSums[i - 1];
		else if (OPERATOR == MINIMUM)
			blockSums[i] = blockSums[i] > blockSums[i - 1] ? blockSums[i - 1] : blockSums[i];
		else if (OPERATOR == MAXIMUM)
			blockSums[i] = blockSums[i] < blockSums[i - 1] ? blockSums[i - 1] : blockSums[i];
		//printf("\nblocksums[%d] = %lld", i, blockSums[i]);
	}

	auto elapsed = std::chrono::high_resolution_clock::now() - start;

	parallelTime += ((double)(std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count()))/1000.00;

	hipEventRecord(start2);
	hipMemcpy(devBlockSums, blockSums, allocSize2, hipMemcpyHostToDevice);
	/*serialEnd = clock();
	parallelEnd = clock();
	parallelTime += (((double)(parallelEnd - parallelStart)) / (double)CLOCKS_PER_SEC) * 1000.00;
	serialTime += (double)(serialEnd - serialStart);*/

	addBlockSums <<<blockCount, threadsPerBlock>>> (devOutArr, devBlockSums, (long)ARR_SIZE);
	hipEventRecord(stop2);

	hipMemcpy(outArr, devOutArr, allocSize1, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&elapsedTime2, start2, stop2);

	//parallelTime += (double)elapsedTime1 + elapsedTime2;

	if (argc > 1 && (argv[1][0] == 'v' || argv[1][0] == 'V')) {
		int minm = (ARR_SIZE <= 100 ? ARR_SIZE : 100);
		printf("\nFirst %d elements of input and output:-", minm);
		printf("\nInput: [");
		for (int i = 0; i < minm; i++)
			printf("%lld, ", inArr[i]);
		printf("]");

		printf("\nOutput: [");
		for (int i = 0; i < minm; i++)
			printf("%lld, ", outArr[i]);
		printf("]");
	}

	//verifyOutput(linearScanOutput, outArr, ARR_SIZE);

	/*printf("\nScan Operation performed: ");
	if (OPERATOR == ADDITION) printf("Addition");
	else if (OPERATOR == MINIMUM) printf("Minimum");
	else if (OPERATOR == MAXIMUM) printf("Maximum");
	printf("\nInput array size = %lld", ARR_SIZE);
	printf("\nNo of threads per block = %d, no. of blocks = %d\n", threadsPerBlock, blockCount);
	printf("\nTime taken by CPU(linear) scan = %Lf ms", linearTime);
	printf("\nTime taken by GPU(parallel) scan = %Lf ms\n", parallelTime);*/


	free(inArr);
	free(outArr);
	free(blockSums);
	hipFree(devInArr);
	hipFree(devOutArr);
	hipFree(devBlockSums);

	parallelTime += elapsedTime1 + elapsedTime2;

	//printf("\nSerialTime = %Lf ms", (serialTime / (double)CLOCKS_PER_SEC)*1000.00);
	printf("\nOverall parallel program time = %Lf ms", parallelTime);
	printf("\nNo. of blocks = %d", blockCount);
	//printf("\nSerial fraction = %Lf", (serialTime / masterTime));
	return 0;
}